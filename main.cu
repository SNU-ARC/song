#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include "logger.h"
#include "parser_dense.h"
#include "parser.h"
#include "data.h"
#include "kernelgraph.h"
#include "config.h"
#include <chrono>

std::unique_ptr<Data> data;
std::unique_ptr<GraphWrapper> graph; 
int topk = 0;
int display_topk = 1;

void build_callback(idx_t idx,std::vector<std::pair<int,value_t>> point){
	if (idx % 10000 == 0)
		printf("idx: %d, point_size: %d\n", idx, point.size());
	data->add(idx,point);
	graph->add_vertex(idx,point);
}

std::vector<std::vector<std::pair<int,value_t>>> batch_queries;
std::vector<std::vector<idx_t>> results(ACC_BATCH_SIZE); // 1000000
std::vector<std::vector<idx_t>> groundtruths(ACC_BATCH_SIZE); // 1000000
unsigned int groundtruth_dim = 0;

void flush_queries(int set_single_batch) {
	unsigned int topk_hit = 0;

	int old_batch_size = batch_queries.size();
	int new_batch_size = old_batch_size;
	int new_num_iter = 1;
	
	if (set_single_batch) {
		new_batch_size = 1;
		new_num_iter = old_batch_size;
	}

	fprintf(stderr, "Original Batch size : %d, New batch size : %d, Num iter : %d\n", old_batch_size, new_batch_size, new_num_iter);

	for (int q = 0; q < new_num_iter; q++) {
		std::vector<std::vector<std::pair<int, value_t>>> batch_queries_size_new;

		for (int i = 0; i < new_batch_size; i++) {
			batch_queries_size_new.push_back(batch_queries[q * new_batch_size + i]);
		}

		fprintf(stderr, "Batch size after insert query : %d\n", batch_queries_size_new.size());

		std::vector<std::vector<idx_t>> results_size_new(new_batch_size);

		graph->search_top_k_batch(batch_queries_size_new, topk, results_size_new);

		for (int i = 0; i < new_batch_size; ++i) {
			std::vector<idx_t> result;
			std::vector<idx_t> groundtruth;

			result = results_size_new[i];
			groundtruth = groundtruths[q * new_batch_size + i];

			for (int j = 0; j < display_topk; j++) {
				for (int k = 0; k < display_topk; k++) {
					if (result[j] == groundtruth[k]) {
						topk_hit++;
						break;
					}
				}
			}
		}
	}
	fprintf(stderr, "Recall@%d: %lf\%\n", display_topk, (float)topk_hit / (new_batch_size * new_num_iter * display_topk) * 100);

	WarpAStarAccelerator::free_all();
	batch_queries.clear();
}

void query_callback(idx_t idx,std::vector<std::pair<int,value_t>> point){
	batch_queries.push_back(point);
	// Uncomment the following lines to have a finer granularity batch processing
	//if(batch_queries.size() == ACC_BATCH_SIZE){
	//    flush_queries();
	//}
	/////////////////////
}

void load_groundtruth(const char* groundtruth_path) {
	auto fp = fopen(groundtruth_path, "rb");
	if (fp == NULL) {
		Logger::log(Logger::ERROR, "File not found at (%s)\n", groundtruth_path);
		exit(1);
	}
	fread(&groundtruth_dim, sizeof(unsigned int), 1, fp);

	fseek(fp, 0, SEEK_END);
	size_t fsize = ftell(fp);
	unsigned int num = (unsigned int)(fsize / (groundtruth_dim + 1) / 4);

	fseek(fp, 0 ,SEEK_SET);
	for (size_t i = 0; i < num; i++) {
		unsigned int value; 
		fseek(fp, 4, SEEK_CUR);
		for (unsigned int j = 0; j < groundtruth_dim; j++) {
			fread(&value, sizeof(unsigned int), 1, fp);
			groundtruths[i].push_back((idx_t)value);
		}
	}
	fclose(fp);
}

void usage(char** argv){
	printf("Usage: %s <build/test> <build_data> <query_data> <search_top_k> <row> <dim> <return_top_k> <l2/ip/cos> <groundtruth>\n",argv[0]);
}

value_t* WarpAStarAccelerator::d_data = NULL;
value_t* WarpAStarAccelerator::d_query = NULL;
idx_t* WarpAStarAccelerator::d_result = NULL;
idx_t* WarpAStarAccelerator::d_graph = NULL;

int main(int argc,char** argv){
	if(argc != 10 && argc != 11){
		usage(argv);
		return 1;
	}
	// You may need to increase this parameter for some new GPUs
	hipDeviceSetLimit(hipLimitMallocHeapSize,800*1024*1024);
	//////////////////////
	size_t row = atoll(argv[5]);
	int dim = atoi(argv[6]);

	display_topk = atoi(argv[7]);
	std::string dist_type = argv[8];
	data = std::unique_ptr<Data>(new Data(row,dim));
	if(dist_type == "l2"){
		graph = std::unique_ptr<GraphWrapper>(new KernelFixedDegreeGraph<0>(data.get())); 
	}else if(dist_type == "ip"){
		graph = std::unique_ptr<GraphWrapper>(new KernelFixedDegreeGraph<1>(data.get())); 
	}else if(dist_type == "cos"){
		graph = std::unique_ptr<GraphWrapper>(new KernelFixedDegreeGraph<2>(data.get())); 
	}else{
		usage(argv);
		return 1;
	}
	std::string mode = std::string(argv[1]);
	topk = atoi(argv[4]);
	if(mode == "build"){
		//std::unique_ptr<ParserDense> build_parser(new ParserDense(argv[2],build_callback));
		std::unique_ptr<Parser> build_parser(new Parser(argv[2], build_callback));
		fprintf(stderr,"Writing the graph and data...");    
		data->dump();
		fprintf(stderr,"...");    
		graph->dump();
		fprintf(stderr,"done\n");    
	}else if(mode == "test"){
		fprintf(stderr,"Loading the graph and data...");    
		data->load();
		fprintf(stderr,"...");    
		graph->load();
		fprintf(stderr,"done\n");    
		load_groundtruth(argv[9]);
		fprintf(stderr,"Loading groundtruth done\n");
		//std::unique_ptr<ParserDense> query_parser(new ParserDense(argv[3],query_callback));
		std::unique_ptr<Parser> query_parser(new Parser(argv[3],query_callback));
		int set_single_batch = atoi(argv[10]);
		flush_queries(set_single_batch);
	}else{
		usage(argv);
		return 1;
	}
	return 0;
}
