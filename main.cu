#include "hip/hip_runtime.h"
#include<stdio.h>
#include<string.h>
#include"logger.h"
#include"parser_dense.h"
#include"parser.h"
#include"data.h"
#include"kernelgraph.h"
#include"config.h"
#include <chrono>

std::unique_ptr<Data> data;
std::unique_ptr<GraphWrapper> graph; 
int topk = 0;
int display_topk = 1;

void build_callback(idx_t idx,std::vector<std::pair<int,value_t>> point){
  if (idx % 10000 == 0)
  printf("idx: %d, point_size: %d\n", idx, point.size());
    data->add(idx,point);
    graph->add_vertex(idx,point);
}

std::vector<std::vector<std::pair<int,value_t>>> batch_queries;
std::vector<std::vector<idx_t>> results(ACC_BATCH_SIZE);
std::vector<std::vector<idx_t>> groundtruths(ACC_BATCH_SIZE);
unsigned int groundtruth_dim = 0;

void flush_queries(){
  results.resize(batch_queries.size());
  const int repeat = 1; // NOTICE: You can repeat multiple times to have an average search performance
//  auto query_start = std::chrono::high_resolution_clock::now();
  for(int i = 0;i < repeat;++i)
    graph->search_top_k_batch(batch_queries,topk,results);
//  auto query_end = std::chrono::high_resolution_clock::now();
//  std::chrono::duration<double> query_diff = query_end - query_start;
//  printf("Search Time: %lf.\n", query_diff.count() / repeat);
//  printf("QPS: %lf.\n", results.size() / (query_diff.count() / repeat));

  unsigned int topk_hit = 0;
  for(int i = 0;i < batch_queries.size();++i){
    auto& result = results[i];
    auto& groundtruth = groundtruths[i];
    for(int j = 0; j < result.size() && j < display_topk; ++j) {
      for (int k = 0; k < display_topk; k++) {
        if (result[j] == groundtruth[k]) {
          topk_hit++;
          break;
        }
      }
    }
//    for(int i = 0;i < result.size() && i < display_topk;++i)
//      printf("%zu ",result[i]);
//    printf("\n");
//    for(int i = 0;i < groundtruth.size() && i < display_topk;++i)
//      printf("%zu ",groundtruth[i]);
//    printf("\n\n");
  }
  printf("Recall@%d: %lf\%\n", display_topk, (float)topk_hit / (batch_queries.size() * display_topk) * 100);
  batch_queries.clear();
}

void query_callback(idx_t idx,std::vector<std::pair<int,value_t>> point){
    batch_queries.push_back(point);
	// Uncomment the following lines to have a finer granularity batch processing
    //if(batch_queries.size() == ACC_BATCH_SIZE){
    //    flush_queries();
    //}
	/////////////////////
}

void load_groundtruth(const char* groundtruth_path) {
  auto fp = fopen(groundtruth_path, "rb");
  if (fp == NULL) {
    Logger::log(Logger::ERROR, "File not found at (%s)\n", groundtruth_path);
    exit(1);
  }
  fread(&groundtruth_dim, sizeof(unsigned int), 1, fp);
  
  fseek(fp, 0, SEEK_END);
  size_t fsize = ftell(fp);
  unsigned int num = (unsigned int)(fsize / (groundtruth_dim + 1) / 4);

  fseek(fp, 0 ,SEEK_SET);
  for (size_t i = 0; i < num; i++) {
    unsigned int value; 
    fseek(fp, 4, SEEK_CUR);
    for (unsigned int j = 0; j < groundtruth_dim; j++) {
      fread(&value, sizeof(unsigned int), 1, fp);
      groundtruths[i].push_back((idx_t)value);
    }
  }
  fclose(fp);
}

void usage(char** argv){
    printf("Usage: %s <build/test> <build_data> <query_data> <search_top_k> <row> <dim> <return_top_k> <l2/ip/cos> <groundtruth>\n",argv[0]);
}

int main(int argc,char** argv){
    if(argc != 10){
        usage(argv);
        return 1;
    }
	// You may need to increase this parameter for some new GPUs
    hipDeviceSetLimit(hipLimitMallocHeapSize,800*1024*1024);
	//////////////////////
	size_t row = atoll(argv[5]);
	int dim = atoi(argv[6]);
	display_topk = atoi(argv[7]);
	std::string dist_type = argv[8];
	data = std::unique_ptr<Data>(new Data(row,dim));
	if(dist_type == "l2"){
		graph = std::unique_ptr<GraphWrapper>(new KernelFixedDegreeGraph<0>(data.get())); 
	}else if(dist_type == "ip"){
		graph = std::unique_ptr<GraphWrapper>(new KernelFixedDegreeGraph<1>(data.get())); 
	}else if(dist_type == "cos"){
		graph = std::unique_ptr<GraphWrapper>(new KernelFixedDegreeGraph<2>(data.get())); 
	}else{
		usage(argv);
		return 1;
	}
    std::string mode = std::string(argv[1]);
    topk = atoi(argv[4]);
    if(mode == "build"){
        //std::unique_ptr<ParserDense> build_parser(new ParserDense(argv[2],build_callback));
        std::unique_ptr<Parser> build_parser(new Parser(argv[2],build_callback));
        fprintf(stderr,"Writing the graph and data...");    
        data->dump();
        fprintf(stderr,"...");    
        graph->dump();
        fprintf(stderr,"done\n");    
    }else if(mode == "test"){
        fprintf(stderr,"Loading the graph and data...");    
        data->load();
        fprintf(stderr,"...");    
        graph->load();
        fprintf(stderr,"done\n");    
        load_groundtruth(argv[9]);
        fprintf(stderr,"Loading groundtruth done\n");
        //std::unique_ptr<ParserDense> query_parser(new ParserDense(argv[3],query_callback));
        std::unique_ptr<Parser> query_parser(new Parser(argv[3],query_callback));
		flush_queries();	
    }else{
        usage(argv);
        return 1;
    }
    return 0;
}
